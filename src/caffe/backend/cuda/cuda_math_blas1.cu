#ifdef USE_CUDA
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <functional>

#include "caffe/backend/cuda/cuda_device.hpp"
#include "caffe/common.hpp"
#include "caffe/backend/backend.hpp"
#include "caffe/backend/vptr.hpp"
#include "caffe/backend/dev_ptr.hpp"
#include "caffe/backend/cuda/caffe_cuda.hpp"
#include "caffe/backend/cuda/cuda_dev_ptr.hpp"

#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

namespace caffe {

void CudaDevice::axpy_float(const uint_tp n, const float alpha,
                            vptr<const float> x, vptr<float> y) {
  CUBLAS_CHECK(hipblasSaxpy(Caffe::cublas_handle(), n, &alpha,
                           x.get_cuda_ptr(), 1, y.get_cuda_ptr(), 1));
}

void CudaDevice::axpy_double(const uint_tp n, const double alpha,
                             vptr<const double> x, vptr<double> y) {
  CUBLAS_CHECK(hipblasDaxpy(Caffe::cublas_handle(), n, &alpha,
                           x.get_cuda_ptr(), 1, y.get_cuda_ptr(), 1));
}

void CudaDevice::axpby_half(const uint_tp n, const half_fp alpha,
                            vptr<const half_fp> x,
                            const half_fp beta,
                            vptr<half_fp> y) {
  this->scal_half(n, beta, y);
  this->axpy_half(n, alpha, x, y);
}

void CudaDevice::axpby_float(const uint_tp n, const float alpha,
                             vptr<const float> x, const float beta,
                             vptr<float> y) {
  this->scal_float(n, beta, y);
  this->axpy_float(n, alpha, x, y);
}

void CudaDevice::axpby_double(const uint_tp n, const double alpha,
                              vptr<const double> x,
                              const double beta, vptr<double> y) {
  this->scal_double(n, beta, y);
  this->axpy_double(n, alpha, x, y);
}

void CudaDevice::scal_float(const uint_tp n, const float alpha,
                            vptr<float> x) {
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(),
                           n, &alpha, x.get_cuda_ptr(), 1));
}

void CudaDevice::scal_double(const uint_tp n, const double alpha,
                             vptr<double> x) {
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), n, &alpha,
                           x.get_cuda_ptr(), 1));
}

template <>
void CudaDevice::scal_str<half_fp>(const int_tp n,
                                            const half_fp alpha,
                                            vptr<half_fp> x,
                                            hipStream_t str) {
#ifdef USE_HALF
  NOT_IMPLEMENTED;  // TODO
#else  // USE_HALF
  NOT_IMPLEMENTED;
#endif  // USE_HALF
}

template <>
void CudaDevice::scal_str<float>(const int_tp n, const float alpha,
                                 vptr<float> x, hipStream_t str) {
  hipStream_t initial_stream;
  CUBLAS_CHECK(hipblasGetStream(Caffe::cublas_handle(), &initial_stream));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), str));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, x.get_cuda_ptr(),
                           1));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), initial_stream));
}

template <>
void CudaDevice::scal_str<double>(const int_tp n, const double alpha,
                                  vptr<double> x, hipStream_t str) {
  hipStream_t initial_stream;
  CUBLAS_CHECK(hipblasGetStream(Caffe::cublas_handle(), &initial_stream));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), str));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), n, &alpha, x.get_cuda_ptr(),
                           1));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), initial_stream));
}

void CudaDevice::dot_float(const uint_tp n, vptr<const float> x,
                           vptr<const float> y, float* out) {
  CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(), n, x.get_cuda_ptr(), 1,
                          y.get_cuda_ptr(), 1, out));
}

void CudaDevice::dot_double(const uint_tp n, vptr<const double> x,
                            vptr<const double> y, double* out) {
  CUBLAS_CHECK(hipblasDdot(Caffe::cublas_handle(), n, x.get_cuda_ptr(), 1,
                          y.get_cuda_ptr(), 1, out));
}

void CudaDevice::asum_float(const uint_tp n, vptr<const float> x, float* y) {
  CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(), n, x.get_cuda_ptr(),
                           1, y));
}

void CudaDevice::asum_double(const uint_tp n, vptr<const double> x, double* y) {
  CUBLAS_CHECK(hipblasDasum(Caffe::cublas_handle(), n, x.get_cuda_ptr(),
                           1, y));
}

void CudaDevice::scale_float(const uint_tp n, const float alpha,
                             vptr<const float> x, vptr<float> y) {
  CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), n, x.get_cuda_ptr(), 1,
                           y.get_cuda_ptr(), 1));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, y.get_cuda_ptr(),
                           1));
}

void CudaDevice::scale_double(const uint_tp n, const double alpha,
                              vptr<const double> x, vptr<double> y) {
  CUBLAS_CHECK(hipblasDcopy(Caffe::cublas_handle(), n, x.get_cuda_ptr(), 1,
                           y.get_cuda_ptr(), 1));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), n, &alpha, y.get_cuda_ptr(),
                           1));
}

#endif  // USE_CUDA
}  // namespace caffe
