#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <functional>

#include "caffe/backend/cuda/cuda_device.hpp"
#include "caffe/common.hpp"
#include "caffe/backend/backend.hpp"
#include "caffe/backend/vptr.hpp"
#include "caffe/backend/dev_ptr.hpp"
#include "caffe/backend/cuda/caffe_cuda.hpp"
#include "caffe/backend/cuda/cuda_dev_ptr.hpp"

#ifdef USE_CUDA
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>
#endif  // USE_CUDA

namespace caffe {

#ifdef USE_CUDA

void CudaDevice::gemv_half(const CBLAS_TRANSPOSE trans_a, const uint_tp m,
                           const uint_tp n, const half_float::half alpha,
                           vptr<const half_float::half> a,
                           vptr<const half_float::half> x,
                           const half_float::half beta,
                           vptr<half_float::half> y) {
#ifdef USE_GPU_HALF
  NOT_IMPLEMENTED;  // TODO
#else  // USE_GPU_HALF
  NOT_IMPLEMENTED;
#endif  // USE_GPU_HALF
}

void CudaDevice::gemv_float(const CBLAS_TRANSPOSE trans_a, const uint_tp m,
                            const uint_tp n, const float alpha,
                            vptr<const float> a,
                            vptr<const float> x, const float beta,
                            vptr<float> y) {
  hipblasOperation_t cuTransA =
      (trans_a == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA,
                           n, m, &alpha, a.get_cuda_ptr(),
                           n, x.get_cuda_ptr(), 1, &beta, y.get_cuda_ptr(), 1));
}

void CudaDevice::gemv_double(const CBLAS_TRANSPOSE trans_a, const uint_tp m,
                             const uint_tp n, const double alpha,
                             vptr<const double> a,
                             vptr<const double> x, const double beta,
                             vptr<double> y) {
  hipblasOperation_t cuTransA =
      (trans_a == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Caffe::cublas_handle(), cuTransA,
                           n, m, &alpha, a.get_cuda_ptr(),
                           n, x.get_cuda_ptr(), 1, &beta, y.get_cuda_ptr(), 1));
}


#endif  // USE_CUDA

}
