#include <cmath>
#include <cstdlib>
#include <cstring>
#include <functional>

#include "caffe/backend/cuda/cuda_device.hpp"
#include "caffe/common.hpp"
#include "caffe/backend/backend.hpp"
#include "caffe/backend/vptr.hpp"
#include "caffe/backend/dev_ptr.hpp"
#include "caffe/backend/cuda/caffe_cuda.hpp"
#include "caffe/backend/cuda/cuda_dev_ptr.hpp"

#ifdef USE_CUDA
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>
#endif  // USE_CUDA

namespace caffe {

#ifdef USE_CUDA

void CudaDevice::gemv_half(const CBLAS_TRANSPOSE trans_a, const uint_tp m,
                           const uint_tp n, const half_fp alpha,
                           vptr<const half_fp> a,
                           vptr<const half_fp> x,
                           const half_fp beta,
                           vptr<half_fp> y) {
#ifdef USE_HALF
  NOT_IMPLEMENTED;  // TODO
#else  // USE_HALF
  NOT_IMPLEMENTED;
#endif  // USE_HALF
}

void CudaDevice::gemv_float(const CBLAS_TRANSPOSE trans_a, const uint_tp m,
                            const uint_tp n, const float alpha,
                            vptr<const float> a,
                            vptr<const float> x, const float beta,
                            vptr<float> y) {
  hipblasOperation_t cuTransA =
      (trans_a == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA,
                           n, m, &alpha, a.get_cuda_ptr(),
                           n, x.get_cuda_ptr(), 1, &beta, y.get_cuda_ptr(), 1));
}

void CudaDevice::gemv_double(const CBLAS_TRANSPOSE trans_a, const uint_tp m,
                             const uint_tp n, const double alpha,
                             vptr<const double> a,
                             vptr<const double> x, const double beta,
                             vptr<double> y) {
  hipblasOperation_t cuTransA =
      (trans_a == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Caffe::cublas_handle(), cuTransA,
                           n, m, &alpha, a.get_cuda_ptr(),
                           n, x.get_cuda_ptr(), 1, &beta, y.get_cuda_ptr(), 1));
}


#endif  // USE_CUDA

}
