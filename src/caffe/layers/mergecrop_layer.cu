#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

#ifdef USE_CUDA
template<typename Dtype>
__global__ void CopyForward(const int nthreads, const Dtype* bottom_a,
                            const Dtype* bottom_b, Dtype* top, int num,
                            int channels_a, int channels_b, int height_a,
                            int width_a, int height_b, int width_b) {

  CUDA_KERNEL_LOOP(index, nthreads)
  {

    int pad_h = (height_b - height_a) / 2;
    int pad_w = (width_b - width_a) / 2;

    int batch_id = index / (channels_a * channels_b * height_a * width_a);

    int bottom_id = ((index
        - batch_id * channels_a * channels_b * height_a * width_a)
        / (channels_a * height_a * width_a)) % 2;

    int h = ((index / width_a) % height_a);
    int w = (index % width_a);

    if (bottom_id == 0) {
      int channel_id = (index / ((width_a * height_a)) % channels_a);
      int aidx = ((((batch_id) * channels_a + channel_id) * height_a + h)
          * width_a + w);
      top[index] = bottom_a[aidx];
    } else {
      int channel_id = (index / ((width_a * height_a)) % channels_b);
      int bidx =
          ((((batch_id) * channels_b + channel_id) * height_a + h + pad_h)
              * width_a + w + (h * 2 + 1) * pad_w);
      top[index] = bottom_b[bidx];
    }
  }

}

template<typename Dtype>
__global__ void CopyBackward(const int nthreads, Dtype* bottom_a,
                             const Dtype* top, int num, int channels_a,
                             int channels_b, int height_a, int width_a,
                             int height_b, int width_b) {

  CUDA_KERNEL_LOOP(index, nthreads)
  {

    int batch_id = index / (channels_a * channels_b * height_a * width_a);

    int bottom_id = ((index
        - batch_id * channels_a * channels_b * height_a * width_a)
        / (channels_a * height_a * width_a)) % 2;

    int h = ((index / width_a) % height_a);
    int w = (index % width_a);

    if (bottom_id == 0) {
      int channel_id = (index / ((width_a * height_a)) % channels_a);
      int aidx = ((((batch_id) * channels_a + channel_id) * height_a + h)
          * width_a + w);
      bottom_a[aidx] = top[index];
    }
  }

}
#endif // USE_CUDA

template<typename Dtype>
void MergeCropLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                        const vector<Blob<Dtype>*>& top) {

  int count = top[0]->count() * 2;

  const Dtype* bottom_data_a = bottom[0]->gpu_data();
  const Dtype* bottom_data_b = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  int num = bottom[0]->num();

  // All channels of both inputs are copied
  int channels_a = bottom[0]->channels();
  int channels_b = bottom[1]->channels();

  // Width and height of the smaller input, which should be input 0
  int height_a = bottom[0]->height();
  int width_a = bottom[0]->width();

  int height_b = bottom[1]->height();
  int width_b = bottom[1]->width();

  CopyForward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS) (
      count, bottom_data_a, bottom_data_b, top_data, num, channels_a,
      channels_b, height_a, width_a, height_b, width_b);

}

template<typename Dtype>
void MergeCropLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                         const vector<bool>& propagate_down,
                                         const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  int count = top[0]->count() * 2;

  Dtype* bottom_diff_a = bottom[0]->mutable_gpu_diff();
  const Dtype* top_diff = top[0]->gpu_diff();

  int num = bottom[0]->num();

  // All channels of both inputs are copied
  int channels_a = bottom[0]->channels();
  int channels_b = bottom[1]->channels();

  // Width and height of the smaller input, which should be input 0
  int height_a = bottom[0]->height();
  int width_a = bottom[0]->width();

  int height_b = bottom[1]->height();
  int width_b = bottom[1]->width();

  CopyBackward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS) (
      count, bottom_diff_a, top_diff, num, channels_a, channels_b, height_a,
      width_a, height_b, width_b);
}

INSTANTIATE_LAYER_GPU_FUNCS(MergeCropLayer);

}  // namespace caffe
