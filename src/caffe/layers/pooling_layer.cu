#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

#ifdef USE_GREENTEA
#include "caffe/greentea/greentea.hpp"
#include "caffe/greentea/greentea_math_functions.hpp"
#endif  // USE_GREENTEA

namespace caffe {

#ifdef USE_CUDA
template<typename Dtype>
__global__ void MaxPoolForward(const int nthreads,
                               const Dtype* const bottom_data, const int num,
                               const int channels, const int height,
                               const int width, const int pooled_height,
                               const int pooled_width, const int kernel_h,
                               const int kernel_w, const int stride_h,
                               const int stride_w, const int pad_h,
                               const int pad_w, Dtype* const top_data,
                               int* mask, Dtype* top_mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    const int hend = min(hstart + kernel_h, height);
    const int wend = min(wstart + kernel_w, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    Dtype maxval = -FLT_MAX;
    int maxidx = -1;
    const Dtype* const bottom_slice = bottom_data
        + (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        if (bottom_slice[h * width + w] > maxval) {
          maxidx = h * width + w;
          maxval = bottom_slice[maxidx];
        }
      }
    }
    top_data[index] = maxval;
    if (mask) {
      mask[index] = maxidx;
    } else {
      top_mask[index] = maxidx;
    }
  }
}

template<typename Dtype>
__global__ void AvePoolForward(const int nthreads,
                               const Dtype* const bottom_data, const int num,
                               const int channels, const int height,
                               const int width, const int pooled_height,
                               const int pooled_width, const int kernel_h,
                               const int kernel_w, const int stride_h,
                               const int stride_w, const int pad_h,
                               const int pad_w, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + kernel_h, height + pad_h);
    int wend = min(wstart + kernel_w, width + pad_w);
    const int pool_size = (hend - hstart) * (wend - wstart);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, height);
    wend = min(wend, width);
    Dtype aveval = 0;
    const Dtype* const bottom_slice = bottom_data
        + (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        aveval += bottom_slice[h * width + w];
      }
    }
    top_data[index] = aveval / pool_size;
  }
}

template<typename Dtype>
__global__ void StoPoolForwardTrain(const int nthreads,
                                    const Dtype* const bottom_data,
                                    const int num, const int channels,
                                    const int height, const int width,
                                    const int pooled_height,
                                    const int pooled_width, const int kernel_h,
                                    const int kernel_w, const int stride_h,
                                    const int stride_w, Dtype* const rand_idx,
                                    Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    const int hstart = ph * stride_h;
    const int hend = min(hstart + kernel_h, height);
    const int wstart = pw * stride_w;
    const int wend = min(wstart + kernel_w, width);
    Dtype cumsum = 0.;
    const Dtype* const bottom_slice = bottom_data
        + (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_slice[h * width + w];
      }
    }
    const float thres = rand_idx[index] * cumsum;
    // Second pass: get value, and set index.
    cumsum = 0;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_slice[h * width + w];
        if (cumsum >= thres) {
          rand_idx[index] = ((n * channels + c) * height + h) * width + w;
          top_data[index] = bottom_slice[h * width + w];
          return;
        }
      }
    }
  }
}

template<typename Dtype>
__global__ void StoPoolForwardTest(const int nthreads,
                                   const Dtype* const bottom_data,
                                   const int num, const int channels,
                                   const int height, const int width,
                                   const int pooled_height,
                                   const int pooled_width, const int kernel_h,
                                   const int kernel_w, const int stride_h,
                                   const int stride_w, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    const int hstart = ph * stride_h;
    const int hend = min(hstart + kernel_h, height);
    const int wstart = pw * stride_w;
    const int wend = min(wstart + kernel_w, width);
    // We set cumsum to be 0 to avoid divide-by-zero problems
    Dtype cumsum = FLT_MIN;
    Dtype cumvalues = 0.;
    const Dtype* const bottom_slice = bottom_data
        + (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_slice[h * width + w];
        cumvalues += bottom_slice[h * width + w] * bottom_slice[h * width + w];
      }
    }
    top_data[index] = cumvalues / cumsum;
  }
}

template<typename Dtype>
__global__ void MaxPoolBackward(const int nthreads, const Dtype* const top_diff,
                                const int* const mask,
                                const Dtype* const top_mask, const int num,
                                const int channels, const int height,
                                const int width, const int pooled_height,
                                const int pooled_width, const int kernel_h,
                                const int kernel_w, const int stride_h,
                                const int stride_w, const int pad_h,
                                const int pad_w, Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const int phstart =
        (h + pad_h < kernel_h) ? 0 : (h + pad_h - kernel_h) / stride_h + 1;
    const int phend = min((h + pad_h) / stride_h + 1, pooled_height);
    const int pwstart =
        (w + pad_w < kernel_w) ? 0 : (w + pad_w - kernel_w) / stride_w + 1;
    const int pwend = min((w + pad_w) / stride_w + 1, pooled_width);
    Dtype gradient = 0;
    const int offset = (n * channels + c) * pooled_height * pooled_width;
    const Dtype* const top_diff_slice = top_diff + offset;
    if (mask) {
      const int* const mask_slice = mask + offset;
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          if (mask_slice[ph * pooled_width + pw] == h * width + w) {
            gradient += top_diff_slice[ph * pooled_width + pw];
          }
        }
      }
    } else {
      const Dtype* const top_mask_slice = top_mask + offset;
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          if (top_mask_slice[ph * pooled_width + pw] == h * width + w) {
            gradient += top_diff_slice[ph * pooled_width + pw];
          }
        }
      }
    }
    bottom_diff[index] = gradient;
  }
}

template<typename Dtype>
__global__ void AvePoolBackward(const int nthreads, const Dtype* const top_diff,
                                const int num, const int channels,
                                const int height, const int width,
                                const int pooled_height, const int pooled_width,
                                const int kernel_h, const int kernel_w,
                                const int stride_h, const int stride_w,
                                const int pad_h, const int pad_w,
                                Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width + pad_w;
    const int h = (index / width) % height + pad_h;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    Dtype gradient = 0;
    const Dtype* const top_diff_slice = top_diff
        + (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int hstart = ph * stride_h - pad_h;
        int wstart = pw * stride_w - pad_w;
        int hend = min(hstart + kernel_h, height + pad_h);
        int wend = min(wstart + kernel_w, width + pad_w);
        int pool_size = (hend - hstart) * (wend - wstart);
        gradient += top_diff_slice[ph * pooled_width + pw] / pool_size;
      }
    }
    bottom_diff[index] = gradient;
  }
}

template<typename Dtype>
__global__ void StoPoolBackward(const int nthreads, const Dtype* const rand_idx,
                                const Dtype* const top_diff, const int num,
                                const int channels, const int height,
                                const int width, const int pooled_height,
                                const int pooled_width, const int kernel_h,
                                const int kernel_w, const int stride_h,
                                const int stride_w, Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    Dtype gradient = 0;
    const Dtype* const rand_idx_slice = rand_idx
        + (n * channels + c) * pooled_height * pooled_width;
    const Dtype* const top_diff_slice = top_diff
        + (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        gradient +=
            top_diff_slice[ph * pooled_width + pw]
                * (index
            == static_cast<int>(rand_idx_slice[ph * pooled_width + pw]));
      }
    }
    bottom_diff[index] = gradient;
  }
}

template<typename Dtype>
__global__ void MaxPoolForward(const int nthreads, const Dtype* bottom_data,
                               const int num, const int channels,
                               const int height, const int width,
                               const int pooled_height, const int pooled_width,
                               const int kernel_h, const int kernel_w,
                               const int ext_kernel_h, const int ext_kernel_w,
                               const int stride_h, const int stride_w,
                               const int kstride_h, const int kstride_w,
                               const int pad_h, const int pad_w,
                               Dtype* top_data, int* mask, Dtype* top_mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + ext_kernel_h, height);
    int wend = min(wstart + ext_kernel_w, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    Dtype maxval = -FLT_MAX;
    int maxidx = -1;
    bottom_data += (n * channels + c) * height * width;
    for (int h = hstart; h < hend; h += kstride_h) {
      for (int w = wstart; w < wend; w += kstride_w) {
        if (bottom_data[h * width + w] > maxval) {
          maxidx = h * width + w;
          maxval = bottom_data[maxidx];
        }
      }
    }
    top_data[index] = maxval;
    if (mask) {
      mask[index] = maxidx;
    } else {
      top_mask[index] = maxidx;
    }
  }
}

template<typename Dtype>
__global__ void AvePoolForward(const int nthreads, const Dtype* bottom_data,
                               const int num, const int channels,
                               const int height, const int width,
                               const int pooled_height, const int pooled_width,
                               const int kernel_h, const int kernel_w,
                               const int ext_kernel_h, const int ext_kernel_w,
                               const int stride_h, const int stride_w,
                               const int kstride_h, const int kstride_w,
                               const int pad_h, const int pad_w,
                               Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + ext_kernel_h, height + pad_h);
    int wend = min(wstart + ext_kernel_w, width + pad_w);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, height);
    wend = min(wend, width);
    Dtype aveval = 0;
    bottom_data += (n * channels + c) * height * width;
    int pool_size = 0;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        aveval += bottom_data[h * width + w];
        ++pool_size;
      }
    }
    top_data[index] = aveval / pool_size;
  }
}

template<typename Dtype>
__global__ void StoPoolForwardTrain(const int nthreads,
                                    const Dtype* bottom_data, const int num,
                                    const int channels, const int height,
                                    const int width, const int pooled_height,
                                    const int pooled_width, const int kernel_h,
                                    const int kernel_w, const int ext_kernel_h,
                                    const int ext_kernel_w, const int stride_h,
                                    const int stride_w, const int kstride_h,
                                    const int kstride_w, Dtype* rand_idx,
                                    Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h;
    int hend = min(hstart + ext_kernel_h, height);
    int wstart = pw * stride_w;
    int wend = min(wstart + ext_kernel_w, width);
    Dtype cumsum = 0.;
    bottom_data += (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; h += kstride_h) {
      for (int w = wstart; w < wend; w += kstride_w) {
        cumsum += bottom_data[h * width + w];
      }
    }
    float thres = rand_idx[index] * cumsum;
    // Second pass: get value, and set index.
    cumsum = 0;
    for (int h = hstart; h < hend; h += kstride_h) {
      for (int w = wstart; w < wend; w += kstride_w) {
        cumsum += bottom_data[h * width + w];
        if (cumsum >= thres) {
          rand_idx[index] = ((n * channels + c) * height + h) * width + w;
          top_data[index] = bottom_data[h * width + w];
          return;
        }
      }
    }
  }
}

template<typename Dtype>
__global__ void StoPoolForwardTest(const int nthreads, const Dtype* bottom_data,
                                   const int num, const int channels,
                                   const int height, const int width,
                                   const int pooled_height,
                                   const int pooled_width, const int kernel_h,
                                   const int kernel_w, const int ext_kernel_h,
                                   const int ext_kernel_w, const int stride_h,
                                   const int stride_w, const int kstride_h,
                                   const int kstride_w, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h;
    int hend = min(hstart + ext_kernel_h, height);
    int wstart = pw * stride_w;
    int wend = min(wstart + ext_kernel_w, width);
    // We set cumsum to be 0 to avoid divide-by-zero problems
    Dtype cumsum = FLT_MIN;
    Dtype cumvalues = 0.;
    bottom_data += (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; h += kstride_h) {
      for (int w = wstart; w < wend; w += kstride_w) {
        cumsum += bottom_data[h * width + w];
        cumvalues += bottom_data[h * width + w] * bottom_data[h * width + w];
      }
    }
    top_data[index] = cumvalues / cumsum;
  }
}

template<typename Dtype>
__global__ void MaxPoolBackward(const int nthreads, const Dtype* top_diff,
                                const int* mask, const Dtype* top_mask,
                                const int num, const int channels,
                                const int height, const int width,
                                const int pooled_height, const int pooled_width,
                                const int kernel_h, const int kernel_w,
                                const int ext_kernel_h, const int ext_kernel_w,
                                const int stride_h, const int stride_w,
                                const int kstride_h, const int kstride_w,
                                const int pad_h, const int pad_w,
                                Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;

    int pooled_height_1 = pooled_height - 1;
    int pooled_width_1 = pooled_width - 1;
    int phstart = (h < ext_kernel_h) ? h % kstride_h : (h - ext_kernel_h) + 1;
    int phend =
        (h >= pooled_height) ?
            pooled_height_1 - (pooled_height_1 - phstart) % kstride_h : h;
    int pwstart = (w < ext_kernel_w) ? w % kstride_w : (w - ext_kernel_w) + 1;
    int pwend =
        (w >= pooled_width) ?
            pooled_width_1 - (pooled_width_1 - pwstart) % kstride_w : w;

    Dtype gradient = 0;
    int offset = (n * channels + c) * pooled_height * pooled_width;
    top_diff += offset;
    if (mask) {
      mask += offset;
      for (int ph = phstart; ph <= phend; ph += kstride_h) {
        for (int pw = pwstart; pw <= pwend; pw += kstride_w) {
          if (mask[ph * pooled_width + pw] == h * width + w) {
            gradient += top_diff[ph * pooled_width + pw];
          }
        }
      }
    } else {
      top_mask += offset;
      for (int ph = phstart; ph <= phend; ph += kstride_h) {
        for (int pw = pwstart; pw <= pwend; pw += kstride_w) {
          if (top_mask[ph * pooled_width + pw] == h * width + w) {
            gradient += top_diff[ph * pooled_width + pw];
          }
        }
      }
    }
    bottom_diff[index] = gradient;
  }
}

template<typename Dtype>
__global__ void MaxPoolNDForward(const int n, const int num_axes,
                                 const Dtype* bottom_data,
                                 const int channels, const int* size,
                                 const int* pooled_size, const int* kernel_size,
                                 const int* ext_kernel_size, const int* stride,
                                 const int* kstride, const int* pad,
                                 Dtype* top_data, int* mask, Dtype* top_mask) {
  int d_idx[6];  // NOLINT(runtime/arrays)
  int d_start[6];  // NOLINT(runtime/arrays)
  int d_end[6];  // NOLINT(runtime/arrays)
  int d_iter[6];  // NOLINT(runtime/arrays)
  int i;

  CUDA_KERNEL_LOOP(index, n) {
    int offset = 1;
    int num = index;
    for (i = num_axes - 1; i >= 0; --i) {
      d_idx[i] = index % pooled_size[i];
      d_start[i] = d_idx[i] * stride[i] - pad[i];
      d_end[i] = min(d_start[i] + ext_kernel_size[i], size[i]);
      d_start[i] = max(d_start[i], 0);
      num /= pooled_size[i];
      offset *= size[i];
      d_iter[i] = d_start[i];

      if (d_start[i] >= d_end[i]) {
        top_data[index] = -FLT_MAX;
        if (mask) {
          mask[index] = -1;
        } else {
          top_mask[index] = -1;
        }
        return;
      }
    }
    int chan = num % channels;
    num /= channels;
    offset *= (num * channels + chan);

    Dtype maxval = -FLT_MAX;
    int maxidx = -1;
    int final_offset = 0;

    bool incremented;
    do {
      final_offset = offset;
      int size_prod = 1;
      for (i = num_axes - 1; i >= 0; --i) {
        final_offset += d_iter[i] * size_prod;
        size_prod *= size[i];
      }

      if (bottom_data[final_offset] > maxval) {
        maxidx = final_offset;
        maxval = bottom_data[maxidx];
      }

      incremented = false;
      for (i = num_axes - 1; i >= 0; --i) {
        if (d_iter[i] >= d_end[i] - kstride[i]) {
          d_iter[i] = d_start[i];
        } else {
          d_iter[i] += kstride[i];
          incremented = true;
          break;
        }
      }
    } while (incremented);

    top_data[index] = maxval;
    if (mask) {
      mask[index] = maxidx;
    } else {
      top_mask[index] = maxidx;
    }
  }
}

template<typename Dtype>
__global__ void MaxPoolNDBackward(const int n, const int num_axes,
                                  const Dtype* top_diff, const int* mask,
                                  const Dtype* top_mask,
                                  const int channels, const int* size,
                                  const int* pooled_size,
                                  const int* kernel_size,
                                  const int* ext_kernel_size, const int* stride,
                                  const int* kstride, const int* pad,
                                  Dtype* bottom_diff) {
  int d_idx[6];  // NOLINT(runtime/arrays)
  int d_start[6];  // NOLINT(runtime/arrays)
  int d_end[6];  // NOLINT(runtime/arrays)
  int d_iter[6];  // NOLINT(runtime/arrays)
  int i;

  CUDA_KERNEL_LOOP(index, n) {
    // find out the local index
    // find out the local offset
    int offset = 1;
    int num = index;
    for (i = num_axes - 1; i >= 0; --i) {
      d_idx[i] = num % size[i];
      d_start[i] = (d_idx[i] < ext_kernel_size[i]) ?
          d_idx[i] % kstride[i] : (d_idx[i] - ext_kernel_size[i]) + 1;
      d_end[i] = (d_idx[i] >= pooled_size[i]) ?
          (pooled_size[i] - 1) - (pooled_size[i] - 1 - d_start[i]) %
          kstride[i] : d_idx[i];
      num /= size[i];
      offset *= pooled_size[i];
      d_iter[i] = d_start[i];

      if (d_start[i] > d_end[i]) {
        bottom_diff[index] = 0;
        return;
      }
    }
    int chan = num % channels;
    num /= channels;
    offset *= (num * channels + chan);

    Dtype gradient = 0;
    int final_offset = 0;
    int im_offset = 0;

    bool incremented;
    do {
      final_offset = offset;
      im_offset = 0;
      int size_prod = 1;
      int pooled_size_prod = 1;
      for (i = num_axes - 1; i >= 0; --i) {
        final_offset += d_iter[i] * pooled_size_prod;
        im_offset += d_idx[i] * size_prod;
        size_prod *= size[i];
        pooled_size_prod *= pooled_size[i];
      }

      if (mask) {
        if (mask[final_offset] == im_offset) {
          gradient += top_diff[final_offset];
        }
      } else {
        if (top_mask[final_offset] == im_offset) {
          gradient += top_diff[final_offset];
        }
      }

      incremented = false;
      for (i = num_axes - 1; i >= 0; --i) {
        if (d_iter[i] > d_end[i] - kstride[i]) {
          d_iter[i] = d_start[i];
        } else {
          d_iter[i] += kstride[i];
          incremented = true;
          break;
        }
      }
    } while (incremented);
    bottom_diff[index] = gradient;
  }
}
#endif  // USE_CUDA



template<typename Dtype>
void PoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                        const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  // We'll output the mask to top[1] if it's of size >1.
  const bool use_top_mask = top.size() > 1;
  int* mask = NULL;
  Dtype* top_mask = NULL;

  if (this->device_context_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA

    if(num_spatial_axes_ == 2) {

      int kernel_h_ = kernel_shape_.cpu_data()[0];
      int kernel_w_ = kernel_shape_.cpu_data()[1];
      int stride_h_ = stride_.cpu_data()[0];
      int stride_w_ = stride_.cpu_data()[1];
      int pad_h_ = pad_.cpu_data()[0];
      int pad_w_ = pad_.cpu_data()[1];
      int kstride_h_ = kstride_.cpu_data()[0];
      int kstride_w_ = kstride_.cpu_data()[1];
      int height_ = size_.cpu_data()[0];
      int width_ = size_.cpu_data()[1];
      int pooled_height_ = pooled_size_.cpu_data()[0];
      int pooled_width_ = pooled_size_.cpu_data()[1];
      int ext_kernel_h = ext_kernel_shape_.cpu_data()[0];
      int ext_kernel_w = ext_kernel_shape_.cpu_data()[0];

      // 2D case
      if(use_skernel_) {
        // 2D-SK case
        switch (this->layer_param_.pooling_param().pool()) {
          case PoolingParameter_PoolMethod_MAX:
            if (use_top_mask) {
              top_mask = top[1]->mutable_gpu_data();
            } else {
              mask = max_idx_.mutable_gpu_data();
            }
            // NOLINT_NEXT_LINE(whitespace/operators)
            MaxPoolForward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                                              CAFFE_CUDA_NUM_THREADS)(
                count, bottom_data, bottom[0]->num(), channels_,
                height_, width_, pooled_height_, pooled_width_, kernel_h_,
                kernel_w_, ext_kernel_h, ext_kernel_w,
                stride_h_, stride_w_, kstride_h_, kstride_w_,
                pad_h_, pad_w_, top_data,
                mask, top_mask);
            break;
          case PoolingParameter_PoolMethod_AVE:
            // NOLINT_NEXT_LINE(whitespace/operators)
            AvePoolForward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                                              CAFFE_CUDA_NUM_THREADS)(
                count, bottom_data, bottom[0]->num(), channels_,
                height_, width_, pooled_height_, pooled_width_, kernel_h_,
                kernel_w_, ext_kernel_h, ext_kernel_w,
                stride_h_, stride_w_, kstride_h_, kstride_w_,
                pad_h_, pad_w_, top_data);
            break;
          case PoolingParameter_PoolMethod_STOCHASTIC:
            if (this->phase_ == caffe::TRAIN) {
              // We need to create the random index as well.
              caffe_gpu_rng_uniform(count, Dtype(0), Dtype(1),
                                    rand_idx_.mutable_gpu_data());
              // NOLINT_NEXT_LINE(whitespace/operators)
              StoPoolForwardTrain<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                  CAFFE_CUDA_NUM_THREADS)(
                  count, bottom_data, bottom[0]->num(), channels_,
                  height_, width_, pooled_height_, pooled_width_, kernel_h_,
                  kernel_w_, ext_kernel_h, ext_kernel_w,
                  stride_h_, stride_w_, kstride_h_, kstride_w_,
                  rand_idx_.mutable_gpu_data(), top_data);
            } else {
              // NOLINT_NEXT_LINE(whitespace/operators)
              StoPoolForwardTest<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                  CAFFE_CUDA_NUM_THREADS)(
                  count, bottom_data, bottom[0]->num(), channels_,
                  height_, width_, pooled_height_, pooled_width_, kernel_h_,
                  kernel_w_, ext_kernel_h, ext_kernel_w,
                  stride_h_, stride_w_, kstride_h_, kstride_w_, top_data);
            }
            break;
          default: {
            LOG(FATAL)<< "Unknown pooling method.";
          }
        }
        CUDA_POST_KERNEL_CHECK;
      } else {
        // 2D case
        switch (this->layer_param_.pooling_param().pool()) {
          case PoolingParameter_PoolMethod_MAX:
            if (use_top_mask) {
              top_mask = top[1]->mutable_gpu_data();
            } else {
              mask = max_idx_.mutable_gpu_data();
            }
            // NOLINT_NEXT_LINE(whitespace/operators)
            MaxPoolForward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                                              CAFFE_CUDA_NUM_THREADS)(
                count, bottom_data, bottom[0]->num(), channels_,
                height_, width_, pooled_height_, pooled_width_, kernel_h_,
                kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data,
                mask, top_mask);
            break;
          case PoolingParameter_PoolMethod_AVE:
            // NOLINT_NEXT_LINE(whitespace/operators)
            AvePoolForward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                                              CAFFE_CUDA_NUM_THREADS)(
                count, bottom_data, bottom[0]->num(), channels_,
                height_, width_, pooled_height_, pooled_width_, kernel_h_,
                kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data);
            break;
          case PoolingParameter_PoolMethod_STOCHASTIC:
            if (this->phase_ == TRAIN) {
              // We need to create the random index as well.
              caffe_gpu_rng_uniform(count, Dtype(0), Dtype(1),
                                    rand_idx_.mutable_gpu_data());
              // NOLINT_NEXT_LINE(whitespace/operators)
              StoPoolForwardTrain<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                  CAFFE_CUDA_NUM_THREADS)(
                  count, bottom_data, bottom[0]->num(), channels_,
                  height_, width_, pooled_height_, pooled_width_, kernel_h_,
                  kernel_w_, stride_h_, stride_w_,
                  rand_idx_.mutable_gpu_data(), top_data);
            } else {
              // NOLINT_NEXT_LINE(whitespace/operators)
              StoPoolForwardTest<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                  CAFFE_CUDA_NUM_THREADS)(
                  count, bottom_data, bottom[0]->num(), channels_,
                  height_, width_, pooled_height_, pooled_width_, kernel_h_,
                  kernel_w_, stride_h_, stride_w_, top_data);
            }
            break;
          default: {
            LOG(FATAL)<< "Unknown pooling method.";
          }
        }
        CUDA_POST_KERNEL_CHECK;
      }
    } else {
      switch (this->layer_param_.pooling_param().pool()) {
        case PoolingParameter_PoolMethod_MAX:
          if (use_top_mask) {
            top_mask = top[1]->mutable_gpu_data();
          } else {
            mask = max_idx_.mutable_gpu_data();
          }
          // NOLINT_NEXT_LINE(whitespace/operators)
          MaxPoolNDForward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                                              CAFFE_CUDA_NUM_THREADS)(
              count, num_spatial_axes_, bottom_data,
              channels_, size_.gpu_data(), pooled_size_.gpu_data(),
              kernel_shape_.gpu_data(), ext_kernel_shape_.gpu_data(),
              stride_.gpu_data(), kstride_.gpu_data(), pad_.gpu_data(),
              top_data, mask, top_mask);
          break;
        default: {
          LOG(FATAL)<< "Unknown pooling method.";
        }
      }
    }
    CUDA_POST_KERNEL_CHECK;

#endif  // USE_CUDA
  } else {
#ifdef USE_GREENTEA
    viennacl::ocl::context &ctx = viennacl::ocl::get_context(
        this->device_context_->id());
    viennacl::ocl::program &program = Caffe::Get().GetDeviceProgram(
        this->device_context_->id());

    if(num_spatial_axes_ == 2) {
      int kernel_h_ = kernel_shape_.cpu_data()[0];
      int kernel_w_ = kernel_shape_.cpu_data()[1];
      int stride_h_ = stride_.cpu_data()[0];
      int stride_w_ = stride_.cpu_data()[1];
      int pad_h_ = pad_.cpu_data()[0];
      int pad_w_ = pad_.cpu_data()[1];
      int kstride_h_ = kstride_.cpu_data()[0];
      int kstride_w_ = kstride_.cpu_data()[1];
      int height_ = size_.cpu_data()[0];
      int width_ = size_.cpu_data()[1];
      int pooled_height_ = pooled_size_.cpu_data()[0];
      int pooled_width_ = pooled_size_.cpu_data()[1];
      int ext_kernel_h = ext_kernel_shape_.cpu_data()[0];
      int ext_kernel_w = ext_kernel_shape_.cpu_data()[0];

      // 2D case
      if(use_skernel_) {
        // 2D-SK case
        switch (this->layer_param_.pooling_param().pool()) {
          case PoolingParameter_PoolMethod_MAX: {
            if (use_top_mask) {
              top_mask = top[1]->mutable_gpu_data();
            } else {
              mask = max_idx_.mutable_gpu_data();
            }
            viennacl::ocl::kernel &oclk_max_pool_forward = program.get_kernel(
                CL_KERNEL_SELECT("max_pool_forward_sk"));
            viennacl::ocl::enqueue(
                oclk_max_pool_forward(count,
                    WrapHandle((cl_mem) bottom_data, &ctx),
                    bottom[0]->num(), channels_, height_, width_,
                    pooled_height_, pooled_width_, kernel_h_,
                    kernel_w_, ext_kernel_h, ext_kernel_w,
                    stride_h_, stride_w_, kstride_h_, kstride_w_,
                    pad_h_, pad_w_,
                    WrapHandle((cl_mem) top_data, &ctx),
                    mask == NULL ? 0 : 1,
                    WrapHandle((cl_mem) mask, &ctx),
                    WrapHandle((cl_mem) top_mask, &ctx)),
                ctx.get_queue());
          }
          break;
          case PoolingParameter_PoolMethod_AVE: {
            viennacl::ocl::kernel &oclk_ave_pool_forward = program.get_kernel(
                CL_KERNEL_SELECT("ave_pool_forward_sk"));
            viennacl::ocl::enqueue(
                oclk_ave_pool_forward(count,
                    WrapHandle((cl_mem) bottom_data, &ctx),
                    bottom[0]->num(), channels_,
                    height_, width_, pooled_height_, pooled_width_, kernel_h_,
                    kernel_w_, ext_kernel_h, ext_kernel_w,
                    stride_h_, stride_w_, kstride_h_, kstride_w_,
                    pad_h_, pad_w_, WrapHandle((cl_mem)top_data, &ctx)),
                ctx.get_queue());
          }
          break;
          case PoolingParameter_PoolMethod_STOCHASTIC: {
            if (this->phase_ == caffe::TRAIN) {
              // We need to create the random index as well.
              greentea_gpu_rng_uniform(this->device_context_->id(), count,
                  Dtype(0), Dtype(1),
                  (cl_mem)(rand_idx_.mutable_gpu_data()), 0);

              viennacl::ocl::kernel &oclk_sto_pool_forward = program.get_kernel(
                  CL_KERNEL_SELECT("sto_pool_forward_train_sk"));
              viennacl::ocl::enqueue(
                  oclk_sto_pool_forward(count,
                      WrapHandle((cl_mem)bottom_data, &ctx),
                      bottom[0]->num(), channels_,
                      height_, width_, pooled_height_, pooled_width_, kernel_h_,
                      kernel_w_, ext_kernel_h, ext_kernel_w,
                      stride_h_, stride_w_, kstride_h_, kstride_w_,
                      WrapHandle((cl_mem)(rand_idx_.mutable_gpu_data()), &ctx),
                      WrapHandle((cl_mem)(top_data), &ctx)),
                  ctx.get_queue());
            } else {
              viennacl::ocl::kernel &oclk_sto_pool_forward = program.get_kernel(
                  CL_KERNEL_SELECT("sto_pool_forward_test_sk"));
              viennacl::ocl::enqueue(
                  oclk_sto_pool_forward(count,
                      WrapHandle((cl_mem)bottom_data, &ctx),
                      bottom[0]->num(), channels_,
                      height_, width_, pooled_height_, pooled_width_, kernel_h_,
                      kernel_w_, ext_kernel_h, ext_kernel_w,
                      stride_h_, stride_w_, kstride_h_, kstride_w_,
                      WrapHandle((cl_mem)top_data, &ctx)),
                  ctx.get_queue());
            }
          }
          break;
          default: {
            LOG(FATAL)<< "Unknown pooling method.";
          }
        }
      } else {
        // 2D case
        switch (this->layer_param_.pooling_param().pool()) {
          case PoolingParameter_PoolMethod_MAX: {
            if (use_top_mask) {
              top_mask = top[1]->mutable_gpu_data();
            } else {
              mask = max_idx_.mutable_gpu_data();
            }
            viennacl::ocl::kernel &oclk_max_pool_forward = program.get_kernel(
                CL_KERNEL_SELECT("max_pool_forward"));
            viennacl::ocl::enqueue(
                oclk_max_pool_forward(count, WrapHandle((cl_mem) bottom_data, &ctx),
                    bottom[0]->num(), channels_, height_, width_,
                    pooled_height_, pooled_width_, kernel_h_,
                    kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
                    WrapHandle((cl_mem) top_data, &ctx),
                    mask == NULL ? 0 : 1,
                    WrapHandle((cl_mem) mask, &ctx),
                    WrapHandle((cl_mem) top_mask, &ctx)),
                ctx.get_queue());
          }
          break;
          case PoolingParameter_PoolMethod_AVE: {
            viennacl::ocl::kernel &oclk_ave_pool_forward = program.get_kernel(
                CL_KERNEL_SELECT("ave_pool_forward"));
            viennacl::ocl::enqueue(
                oclk_ave_pool_forward(count,
                    WrapHandle((cl_mem) bottom_data, &ctx),
                    bottom[0]->num(), channels_,
                    height_, width_, pooled_height_, pooled_width_, kernel_h_,
                    kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
                    WrapHandle((cl_mem)top_data, &ctx)),
                ctx.get_queue());
          }
          break;
          case PoolingParameter_PoolMethod_STOCHASTIC: {
            if (this->phase_ == caffe::TRAIN) {
              // We need to create the random index as well.
              greentea_gpu_rng_uniform(this->device_context_->id(), count,
                  Dtype(0), Dtype(1),
                  (cl_mem)(rand_idx_.mutable_gpu_data()), 0);

              viennacl::ocl::kernel &oclk_sto_pool_forward = program.get_kernel(
                  CL_KERNEL_SELECT("sto_pool_forward_train"));
              viennacl::ocl::enqueue(
                  oclk_sto_pool_forward(count,
                      WrapHandle((cl_mem)bottom_data, &ctx),
                      bottom[0]->num(), channels_,
                      height_, width_, pooled_height_, pooled_width_,
                      kernel_h_, kernel_w_,
                      stride_h_, stride_w_,
                      WrapHandle((cl_mem)(rand_idx_.mutable_gpu_data()), &ctx),
                      WrapHandle((cl_mem)top_data, &ctx)),
                  ctx.get_queue());
            } else {
              viennacl::ocl::kernel &oclk_sto_pool_forward = program.get_kernel(
                  CL_KERNEL_SELECT("sto_pool_forward_test"));
              viennacl::ocl::enqueue(
                  oclk_sto_pool_forward(count,
                      WrapHandle((cl_mem)bottom_data, &ctx),
                      bottom[0]->num(), channels_,
                      height_, width_, pooled_height_,
                      pooled_width_, kernel_h_, kernel_w_,
                      stride_h_, stride_w_, WrapHandle((cl_mem)top_data, &ctx)),
                  ctx.get_queue());
            }
          }
          break;
          default: {
            LOG(FATAL)<< "Unknown pooling method.";
          }
        }
      }
    } else {
      switch (this->layer_param_.pooling_param().pool()) {
        case PoolingParameter_PoolMethod_MAX: {
          if (use_top_mask) {
            top_mask = top[1]->mutable_gpu_data();
          } else {
            mask = max_idx_.mutable_gpu_data();
          }
          viennacl::ocl::kernel &oclk_max_pool_forward = program.get_kernel(
              CL_KERNEL_SELECT("max_pool_forward_nd"));
          viennacl::ocl::enqueue(
              oclk_max_pool_forward(count, num_spatial_axes_,
                  WrapHandle((cl_mem)bottom_data, &ctx),
                  channels_,
                  WrapHandle((cl_mem)(size_.gpu_data()), &ctx),
                  WrapHandle((cl_mem)(pooled_size_.gpu_data()), &ctx),
                  WrapHandle((cl_mem)(kernel_shape_.gpu_data()), &ctx),
                  WrapHandle((cl_mem)(ext_kernel_shape_.gpu_data()), &ctx),
                  WrapHandle((cl_mem)(stride_.gpu_data()), &ctx),
                  WrapHandle((cl_mem)(kstride_.gpu_data()), &ctx),
                  WrapHandle((cl_mem)(pad_.gpu_data()), &ctx),
                  WrapHandle((cl_mem)top_data, &ctx),
                  mask == NULL ? 0 : 1,
                  WrapHandle((cl_mem)mask, &ctx),
                  WrapHandle((cl_mem)top_mask, &ctx)),
              ctx.get_queue());
        }
        break;
        default: {
          LOG(FATAL)<< "Unknown pooling method.";
        }
      }
    }

#endif  // USE_GREENTEA
  }
}

template<typename Dtype>
void PoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                       const vector<bool>& propagate_down,
                                       const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  // We'll output the mask to top[1] if it's of size >1.
  const bool use_top_mask = top.size() > 1;
  const int* mask = NULL;
  const Dtype* top_mask = NULL;

  if (this->device_context_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
    caffe_gpu_set(count, Dtype(0.), bottom_diff);

    if(num_spatial_axes_ == 2) {

      int kernel_h_ = kernel_shape_.cpu_data()[0];
      int kernel_w_ = kernel_shape_.cpu_data()[1];
      int stride_h_ = stride_.cpu_data()[0];
      int stride_w_ = stride_.cpu_data()[1];
      int pad_h_ = pad_.cpu_data()[0];
      int pad_w_ = pad_.cpu_data()[1];
      int kstride_h_ = kstride_.cpu_data()[0];
      int kstride_w_ = kstride_.cpu_data()[1];
      int height_ = size_.cpu_data()[0];
      int width_ = size_.cpu_data()[1];
      int pooled_height_ = pooled_size_.cpu_data()[0];
      int pooled_width_ = pooled_size_.cpu_data()[1];
      int ext_kernel_h = ext_kernel_shape_.cpu_data()[0];
      int ext_kernel_w = ext_kernel_shape_.cpu_data()[0];

      if(use_skernel_) {
        switch (this->layer_param_.pooling_param().pool()) {
          case PoolingParameter_PoolMethod_MAX:
            if (use_top_mask) {
              top_mask = top[1]->gpu_data();
            } else {
              mask = max_idx_.gpu_data();
            }
            // NOLINT_NEXT_LINE(whitespace/operators)
            MaxPoolBackward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                                               CAFFE_CUDA_NUM_THREADS)(
                count, top_diff, mask, top_mask, top[0]->num(), channels_,
                height_, width_, pooled_height_, pooled_width_,
                kernel_h_, kernel_w_, ext_kernel_h, ext_kernel_w,
                stride_h_, stride_w_, kstride_h_, kstride_w_,
                pad_h_, pad_w_,
                bottom_diff);
            break;
          default:
            LOG(FATAL)<<
            "Unknown or unsupported pooling method in Backward_gpu().";
          }
          CUDA_POST_KERNEL_CHECK;
        } else {
          switch (this->layer_param_.pooling_param().pool()) {
            case PoolingParameter_PoolMethod_MAX:
            if (use_top_mask) {
              top_mask = top[1]->gpu_data();
            } else {
              mask = max_idx_.gpu_data();
            }
            // NOLINT_NEXT_LINE(whitespace/operators)
            MaxPoolBackward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                CAFFE_CUDA_NUM_THREADS)(
                count, top_diff, mask, top_mask, top[0]->num(), channels_,
                height_, width_, pooled_height_, pooled_width_,
                kernel_h_, kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
                bottom_diff);
            break;
            case PoolingParameter_PoolMethod_AVE:
            // NOLINT_NEXT_LINE(whitespace/operators)
            AvePoolBackward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                CAFFE_CUDA_NUM_THREADS)(
                count, top_diff, top[0]->num(), channels_,
                height_, width_, pooled_height_, pooled_width_, kernel_h_,
                kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, bottom_diff);
            break;
            case PoolingParameter_PoolMethod_STOCHASTIC:
            // NOLINT_NEXT_LINE(whitespace/operators)
            StoPoolBackward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                CAFFE_CUDA_NUM_THREADS)(
                count, rand_idx_.gpu_data(), top_diff,
                top[0]->num(), channels_, height_, width_, pooled_height_,
                pooled_width_, kernel_h_, kernel_w_, stride_h_, stride_w_,
                bottom_diff);
            break;
            default: {
              LOG(FATAL)<< "Unknown pooling method.";
            }
          }
          CUDA_POST_KERNEL_CHECK;
        }
      } else {
      switch (this->layer_param_.pooling_param().pool()) {
        case PoolingParameter_PoolMethod_MAX:
          if (use_top_mask) {
            top_mask = top[1]->gpu_data();
          } else {
            mask = max_idx_.gpu_data();
          }
          // NOLINT_NEXT_LINE(whitespace/operators)
          MaxPoolNDBackward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
              CAFFE_CUDA_NUM_THREADS)(
              count, num_spatial_axes_, top_diff, mask, top_mask,
              channels_, size_.gpu_data(), pooled_size_.gpu_data(),
              kernel_shape_.gpu_data(), ext_kernel_shape_.gpu_data(),
              stride_.gpu_data(), kstride_.gpu_data(), pad_.gpu_data(),
              bottom_diff);
          break;
        default:
          LOG(FATAL)<<
          "Unknown or unsupported pooling method in Backward_gpu().";
        }
      CUDA_POST_KERNEL_CHECK;
    }
#endif  // USE_CUDA
  } else {
#ifdef USE_GREENTEA
      viennacl::ocl::context &ctx = viennacl::ocl::get_context(
          this->device_context_->id());
      viennacl::ocl::program &program = Caffe::Get().GetDeviceProgram(
          this->device_context_->id());

      greentea_gpu_set(this->device_context_->id(), count, Dtype(0.),
          (cl_mem) bottom_diff, 0);

      if(num_spatial_axes_ == 2) {

        int kernel_h_ = kernel_shape_.cpu_data()[0];
        int kernel_w_ = kernel_shape_.cpu_data()[1];
        int stride_h_ = stride_.cpu_data()[0];
        int stride_w_ = stride_.cpu_data()[1];
        int pad_h_ = pad_.cpu_data()[0];
        int pad_w_ = pad_.cpu_data()[1];
        int kstride_h_ = kstride_.cpu_data()[0];
        int kstride_w_ = kstride_.cpu_data()[1];
        int height_ = size_.cpu_data()[0];
        int width_ = size_.cpu_data()[1];
        int pooled_height_ = pooled_size_.cpu_data()[0];
        int pooled_width_ = pooled_size_.cpu_data()[1];
        int ext_kernel_h = ext_kernel_shape_.cpu_data()[0];
        int ext_kernel_w = ext_kernel_shape_.cpu_data()[0];

        if(use_skernel_) {
          switch (this->layer_param_.pooling_param().pool()) {
            case PoolingParameter_PoolMethod_MAX: {
              if (use_top_mask) {
                top_mask = top[1]->gpu_data();
              } else {
                mask = max_idx_.gpu_data();
              }
              viennacl::ocl::kernel &oclk_max_pool_backward = program.get_kernel(
                  CL_KERNEL_SELECT("max_pool_backward_sk"));
              viennacl::ocl::enqueue(
                  oclk_max_pool_backward(count, WrapHandle((cl_mem) top_diff, &ctx),
                      mask == NULL ? 0 : 1,
                      WrapHandle((cl_mem) mask, &ctx),
                      WrapHandle((cl_mem) top_mask, &ctx),
                      top[0]->num(), channels_, height_, width_,
                      pooled_height_, pooled_width_, kernel_h_,
                      kernel_w_, ext_kernel_h, ext_kernel_w,
                      stride_h_, stride_w_, kstride_h_, kstride_w_,
                      pad_h_, pad_w_,
                      WrapHandle((cl_mem) bottom_diff, &ctx)),
                  ctx.get_queue());
            }
            break;
            default:
            LOG(FATAL)<<
            "Unknown or unsupported pooling method in Backward_gpu().";
          }
        } else {
          switch (this->layer_param_.pooling_param().pool()) {
            case PoolingParameter_PoolMethod_MAX: {
              if (use_top_mask) {
                top_mask = top[1]->gpu_data();
              } else {
                mask = max_idx_.gpu_data();
              }
              viennacl::ocl::kernel &oclk_max_pool_backward = program.get_kernel(
                  CL_KERNEL_SELECT("max_pool_backward"));
              viennacl::ocl::enqueue(
                  oclk_max_pool_backward(count, WrapHandle((cl_mem) top_diff, &ctx),
                      mask == NULL ? 0 : 1,
                      WrapHandle((cl_mem) mask, &ctx),
                      WrapHandle((cl_mem) top_mask, &ctx),
                      top[0]->num(), channels_, height_, width_,
                      pooled_height_, pooled_width_, kernel_h_,
                      kernel_w_, stride_h_, stride_w_, pad_h_,
                      pad_w_,
                      WrapHandle((cl_mem) bottom_diff, &ctx)),
                  ctx.get_queue());
            }
            break;
            case PoolingParameter_PoolMethod_AVE: {
              viennacl::ocl::kernel &oclk_ave_pool_backward = program.get_kernel(
                  CL_KERNEL_SELECT("ave_pool_backward"));
              viennacl::ocl::enqueue(
                  oclk_ave_pool_backward(count, WrapHandle((cl_mem) top_diff, &ctx),
                      top[0]->num(), channels_, height_, width_,
                      pooled_height_, pooled_width_, kernel_h_,
                      kernel_w_, stride_h_, stride_w_, pad_h_,
                      pad_w_,
                      WrapHandle((cl_mem) bottom_diff, &ctx)),
                  ctx.get_queue());
            }
            break;
            case PoolingParameter_PoolMethod_STOCHASTIC: {
              viennacl::ocl::kernel &oclk_sto_pool_backward = program.get_kernel(
                  CL_KERNEL_SELECT("sto_pool_backward"));
              viennacl::ocl::enqueue(
                  oclk_sto_pool_backward(
                      count, WrapHandle((cl_mem) (rand_idx_.gpu_data()), &ctx),
                      WrapHandle((cl_mem) top_diff, &ctx), top[0]->num(), channels_,
                      height_, width_, pooled_height_, pooled_width_, kernel_h_,
                      kernel_w_, stride_h_, stride_w_,
                      WrapHandle((cl_mem) bottom_diff, &ctx)),
                  ctx.get_queue());
            }
            break;
            default: {
              LOG(FATAL)<< "Unknown pooling method.";
            }
          }
        }
      } else {
        switch (this->layer_param_.pooling_param().pool()) {
          case PoolingParameter_PoolMethod_MAX: {
            if (use_top_mask) {
              top_mask = top[1]->gpu_data();
            } else {
              mask = max_idx_.gpu_data();
            }
            viennacl::ocl::kernel &oclk_max_pool_backward = program.get_kernel(
                CL_KERNEL_SELECT("max_pool_backward_nd"));
            viennacl::ocl::enqueue(
                oclk_max_pool_backward(
                    count, num_spatial_axes_, WrapHandle((cl_mem) top_diff, &ctx),
                    mask == NULL ? 0 : 1, WrapHandle((cl_mem) mask, &ctx),
                    WrapHandle((cl_mem) top_mask, &ctx), channels_,
                    WrapHandle((cl_mem) (size_.gpu_data()), &ctx),
                    WrapHandle((cl_mem) (pooled_size_.gpu_data()), &ctx),
                    WrapHandle((cl_mem) (kernel_shape_.gpu_data()), &ctx),
                    WrapHandle((cl_mem) (ext_kernel_shape_.gpu_data()), &ctx),
                    WrapHandle((cl_mem) (stride_.gpu_data()), &ctx),
                    WrapHandle((cl_mem) (kstride_.gpu_data()), &ctx),
                    WrapHandle((cl_mem) (pad_.gpu_data()), &ctx),
                    WrapHandle((cl_mem) bottom_diff, &ctx)),
                ctx.get_queue());
          }
          break;
          default:
          LOG(FATAL)<< "Unknown or unsupported pooling method in Backward_gpu().";
        }
      }
#endif  // USE_GREENTEA
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(PoolingLayer);

}  // namespace caffe
