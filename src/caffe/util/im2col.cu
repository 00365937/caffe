#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <vector>

#include "caffe/common.hpp"
#include "caffe/util/im2col.hpp"

namespace caffe {

#ifdef USE_CUDA
template<typename Dtype>
__global__ void im2col_sk_gpu_kernel(const int n, const Dtype* data_im,
                                     const int height, const int width,
                                     const int kernel_h, const int kernel_w,
                                     const int ext_kernel_h,
                                     const int ext_kernel_w, const int pad_h,
                                     const int pad_w, const int stride_h,
                                     const int stride_w, const int kstride_h,
                                     const int kstride_w, const int height_col,
                                     const int width_col, Dtype* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    int w_out = index % width_col;
    int h_index = index / width_col;
    int h_out = h_index % height_col;
    int channel_in = h_index / height_col;
    int channel_out = channel_in * kernel_h * kernel_w;
    int h_in = h_out * stride_h - pad_h;
    int w_in = w_out * stride_w - pad_w;
    Dtype* data_col_ptr = data_col;
    data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
    const Dtype* data_im_ptr = data_im;
    data_im_ptr += (channel_in * height + h_in) * width + w_in;
    for (int i = 0; i < ext_kernel_h; i += kstride_h) {
      for (int j = 0; j < ext_kernel_w; j += kstride_w) {
        int h = h_in + i;
        int w = w_in + j;
        *data_col_ptr =
            (h >= 0 && w >= 0 && h < height && w < width) ?
                data_im_ptr[i * width + j] : 0;
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

template<typename Dtype>
void im2col_sk_gpu(const Dtype* data_im, const int channels, const int height,
                   const int width, const int kernel_h, const int kernel_w,
                   const int pad_h, const int pad_w, const int stride_h,
                   const int stride_w, const int kstride_h, const int kstride_w,
                   Dtype* data_col) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int ext_kernel_h = (kernel_h - 1) * kstride_h + 1;
  int ext_kernel_w = (kernel_w - 1) * kstride_w + 1;
  int height_col = (height + 2 * pad_h - ext_kernel_h) / stride_h + 1;
  int width_col = (width + 2 * pad_w - ext_kernel_w) / stride_w + 1;
  int num_kernels = channels * height_col * width_col;

  // NOLINT_NEXT_LINE(whitespace/operators)
  im2col_sk_gpu_kernel<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(num_kernels),
      CAFFE_CUDA_NUM_THREADS)(
      num_kernels, data_im, height, width, kernel_h, kernel_w,
      ext_kernel_h, ext_kernel_w, pad_h, pad_w,
      stride_h, stride_w, kstride_h, kstride_w,
      height_col, width_col,
      data_col);
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void im2col_sk_gpu<float>(const float* data_im, const int channels,
                                   const int height, const int width,
                                   const int kernel_h, const int kernel_w,
                                   const int pad_h, const int pad_w,
                                   const int stride_h, const int stride_w,
                                   const int kstride_h, const int kstride_w,
                                   float* data_col);
template void im2col_sk_gpu<double>(const double* data_im, const int channels,
                                    const int height, const int width,
                                    const int kernel_h, const int kernel_w,
                                    const int pad_h, const int pad_w,
                                    const int stride_h, const int stride_w,
                                    const int kstride_h, const int kstride_w,
                                    double* data_col);

template<typename Dtype>
__global__ void im2col_gpu_kernel(const int n, const Dtype* data_im,
                                  const int height, const int width,
                                  const int kernel_h, const int kernel_w,
                                  const int pad_h, const int pad_w,
                                  const int stride_h, const int stride_w,
                                  const int height_col, const int width_col,
                                  Dtype* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    int w_out = index % width_col;
    int h_index = index / width_col;
    int h_out = h_index % height_col;
    int channel_in = h_index / height_col;
    int channel_out = channel_in * kernel_h * kernel_w;
    int h_in = h_out * stride_h - pad_h;
    int w_in = w_out * stride_w - pad_w;
    Dtype* data_col_ptr = data_col;
    data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
    const Dtype* data_im_ptr = data_im;
    data_im_ptr += (channel_in * height + h_in) * width + w_in;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        int h = h_in + i;
        int w = w_in + j;
        *data_col_ptr =
            (h >= 0 && w >= 0 && h < height && w < width) ?
                data_im_ptr[i * width + j] : 0;
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

template<typename Dtype>
void im2col_gpu(const Dtype* data_im, const int channels, const int height,
                const int width, const int kernel_h, const int kernel_w,
                const int pad_h, const int pad_w, const int stride_h,
                const int stride_w, Dtype* data_col) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col = (height + 2 * pad_h - kernel_h) / stride_h + 1;
  int width_col = (width + 2 * pad_w - kernel_w) / stride_w + 1;
  int num_kernels = channels * height_col * width_col;
  // NOLINT_NEXT_LINE(whitespace/operators)
  im2col_gpu_kernel<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(num_kernels),
      CAFFE_CUDA_NUM_THREADS)(
      num_kernels, data_im, height, width, kernel_h, kernel_w, pad_h,
      pad_w, stride_h, stride_w, height_col,
      width_col, data_col);
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void im2col_gpu<float>(const float* data_im, const int channels,
                                const int height, const int width,
                                const int kernel_h, const int kernel_w,
                                const int pad_h, const int pad_w,
                                const int stride_h, const int stride_w,
                                float* data_col);
template void im2col_gpu<double>(const double* data_im, const int channels,
                                 const int height, const int width,
                                 const int kernel_h, const int kernel_w,
                                 const int pad_h, const int pad_w,
                                 const int stride_h, const int stride_w,
                                 double* data_col);

// Support of stride_h and stride_w greater than 1 is not implemented
template<typename Dtype>
__global__ void col2im_sk_gpu_kernel(const int n, const Dtype* data_col,
                                     const int height, const int width,
                                     const int channels, const int patch_h,
                                     const int patch_w, const int ext_patch_h,
                                     const int ext_patch_w, const int pad_h,
                                     const int pad_w, const int stride_h,
                                     const int stride_w, const int kstride_h,
                                     const int kstride_w, const int height_col,
                                     const int width_col, Dtype* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = 0;
    int w = index % width + pad_w;
    int h = (index / width) % height + pad_h;
    int c = index / (width * height);
    // compute the start and end of the output
    int width_col_1 = width_col - 1;
    int height_col_1 = height_col - 1;
    int w_col_start = (w < ext_patch_w) ? w % kstride_w : (w - ext_patch_w) + 1;
    int w_col_end =
        (w >= width_col) ?
            width_col_1 - (width_col_1 - w_col_start) % kstride_w : w;
    int h_col_start = (h < ext_patch_h) ? h % kstride_h : (h - ext_patch_h) + 1;
    int h_col_end =
        (h >= height_col) ?
            height_col_1 - (height_col_1 - h_col_start) % kstride_h : h;
    int w_num = (w - w_col_start) / kstride_w;
    int h_num = (h - h_col_start) / kstride_h;

    int coeff_w_idx = height_col * width_col;
    int coeff_h_idx = patch_w * coeff_w_idx;
    int offset = c * patch_h * coeff_h_idx;
    for (int h_col = h_col_start, h_idx = h_num; h_col <= h_col_end; h_col +=
        kstride_h, --h_idx) {
      for (int w_col = w_col_start, w_idx = w_num; w_col <= w_col_end; w_col +=
          kstride_w, --w_idx) {
        val += data_col[offset + h_idx * coeff_h_idx + w_idx * coeff_w_idx
            + h_col * width_col + w_col];
      }
    }

    data_im[index] = val;
  }
}

template<typename Dtype>
void col2im_sk_gpu(const Dtype* data_col, const int channels, const int height,
                   const int width, const int patch_h, const int patch_w,
                   const int pad_h, const int pad_w, const int stride_h,
                   const int stride_w, const int kstride_h, const int kstride_w,
                   Dtype* data_im) {
  if (stride_w > 1 || stride_h > 1 || pad_h > 0 || pad_w > 0)
    LOG(FATAL)<< "stride greater than 1 or pad greater"
    << " than 0 not tested in col2im_sk_gpu().";
    int ext_patch_h = (patch_h - 1) * kstride_h + 1;
    int ext_patch_w = (patch_w - 1) * kstride_w + 1;
    int height_col = (height + 2 * pad_h - ext_patch_h) / stride_h + 1;
    int width_col = (width + 2 * pad_w - ext_patch_w) / stride_w + 1;
    int num_kernels = channels * height * width;

    col2im_sk_gpu_kernel<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(num_kernels),
        CAFFE_CUDA_NUM_THREADS)(
        num_kernels, data_col, height, width, channels,
        patch_h, patch_w, ext_patch_h, ext_patch_w,
        pad_h, pad_w, stride_h, stride_w, kstride_h, kstride_w,
        height_col, width_col, data_im);
    CUDA_POST_KERNEL_CHECK;
  }

// Explicit instantiation
template void col2im_sk_gpu<float>(const float* data_col, const int channels,
                                   const int height, const int width,
                                   const int patch_h, const int patch_w,
                                   const int pad_h, const int pad_w,
                                   const int stride_h, const int stride_w,
                                   const int kstride_h, const int kstride_w,
                                   float* data_im);
template void col2im_sk_gpu<double>(const double* data_col, const int channels,
                                    const int height, const int width,
                                    const int patch_h, const int patch_w,
                                    const int pad_h, const int pad_w,
                                    const int stride_h, const int stride_w,
                                    const int kstride_h, const int kstride_w,
                                    double* data_im);

template<typename Dtype>
__global__ void col2im_gpu_kernel(const int n, const Dtype* data_col,
                                  const int height, const int width,
                                  const int channels, const int patch_h,
                                  const int patch_w, const int pad_h,
                                  const int pad_w, const int stride_h,
                                  const int stride_w, const int height_col,
                                  const int width_col, Dtype* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = 0;
    int w = index % width + pad_w;
    int h = (index / width) % height + pad_h;
    int c = index / (width * height);

    // compute the start and end of the output
    int w_col_start = (w < patch_w) ? 0 : (w - patch_w) / stride_w + 1;
    int w_col_end = min(w / stride_w + 1, width_col);
    int h_col_start = (h < patch_h) ? 0 : (h - patch_h) / stride_h + 1;
    int h_col_end = min(h / stride_h + 1, height_col);

    // equivalent implementation
    int offset = (c * patch_h * patch_w + h * patch_w + w) * height_col
        * width_col;
    int coeff_h_col = (1 - stride_h * patch_w * height_col) * width_col;
    int coeff_w_col = (1 - stride_w * height_col * width_col);
    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
      }
    }
    data_im[index] = val;
  }
}

template<typename Dtype>
void col2im_gpu(const Dtype* data_col, const int channels, const int height,
                const int width, const int patch_h, const int patch_w,
                const int pad_h, const int pad_w, const int stride_h,
                const int stride_w, Dtype* data_im) {
  int height_col = (height + 2 * pad_h - patch_h) / stride_h + 1;
  int width_col = (width + 2 * pad_w - patch_w) / stride_w + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  col2im_gpu_kernel<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(num_kernels),
      CAFFE_CUDA_NUM_THREADS)(
      num_kernels, data_col, height, width, channels, patch_h, patch_w,
      pad_h, pad_w, stride_h, stride_w,
      height_col, width_col, data_im);
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void col2im_gpu<float>(const float* data_col, const int channels,
                                const int height, const int width,
                                const int patch_h, const int patch_w,
                                const int pad_h, const int pad_w,
                                const int stride_h, const int stride_w,
                                float* data_im);
template void col2im_gpu<double>(const double* data_col, const int channels,
                                 const int height, const int width,
                                 const int patch_h, const int patch_w,
                                 const int pad_h, const int pad_w,
                                 const int stride_h, const int stride_w,
                                 double* data_im);

template<typename Dtype>
__global__ void im2col_ndsk_gpu_kernel(const int n, const int num_axes,
                                       const Dtype* data_im,
                                       const int* im_shape,
                                       const int* col_shape,
                                       const int* kernel_shape, const int* pad,
                                       const int* stride, const int* kstride,
                                       Dtype* data_col) {
  int d_temp[6];  // NOLINT(runtime/arrays)
  int d_iter[6];  // NOLINT(runtime/arrays)
  int i;
  CUDA_KERNEL_LOOP(index, n) {
    // Initialize channel_in, computed in the loop below, with intermediate
    // computations used to compute the spatial indices.
    int channel_in = index;
    int channel_out = 1;
    for (i = num_axes - 1; i >= 0; --i) {
      d_temp[i] = channel_in % col_shape[i + 1];
      channel_in /= col_shape[i + 1];
      channel_out *= kernel_shape[i];
    }
    channel_out *= channel_in;
    int data_col_inc = 1;
    for (i = 0; i < num_axes; ++i) {
      channel_out *= col_shape[i + 1];
      channel_out += d_temp[i];
      d_temp[i] = d_temp[i] * stride[i] - pad[i];
      channel_in *= im_shape[i + 1];
      channel_in += d_temp[i];
      data_col_inc *= col_shape[i + 1];
      d_iter[i] = 0;
    }
    Dtype* data_col_ptr = data_col + channel_out;
    const Dtype* data_im_ptr = data_im + channel_in;
    bool incremented;
    do {
      bool in_range = true;
      for (i = 0; i < num_axes; ++i) {
        const int d_iter_im = d_iter[i] + d_temp[i];
        in_range &= d_iter_im >= 0 && d_iter_im < im_shape[i + 1];
        if (!in_range) {
          break;
        }
      }

      // Write column data
      if (in_range) {
        int data_im_offset = d_iter[0];
        for (i = 1; i < num_axes; ++i) {
          data_im_offset *= im_shape[i + 1];
          data_im_offset += d_iter[i];
        }
        *data_col_ptr = data_im_ptr[data_im_offset];
      } else {
        *data_col_ptr = 0;
      }

      data_col_ptr += data_col_inc;
      incremented = false;
      for (i = num_axes - 1; i >= 0; --i) {
        // Old: const int d_max = kernel_shape[i];
        // New (strided, limit is the external kernel size):
        const int d_max = (kernel_shape[i] - 1) * kstride[i] + 1;
        if (d_iter[i] > d_max - kstride[i]) {
          d_iter[i] = 0;
        } else {  // d_iter[i] <= d_max - kstride[i]
          // Old: ++d_iter[i];
          // New (strided, increment by the stride each time):
          d_iter[i] += kstride[i];
          incremented = true;
          break;
        }
      }  // for (int i = num_axes - 1; i >= 0; --i)
    } while (incremented);  // do
  }  // CUDA_KERNEL_LOOP(index, n)
}

template<typename Dtype>
__global__ void col2im_ndsk_gpu_kernel(const int n, const int num_axes,
                                       const Dtype* data_col,
                                       const int* im_shape,
                                       const int* col_shape,
                                       const int* kernel_shape, const int* pad,
                                       const int* stride, const int* kstride,
                                       Dtype* data_im) {
  int d_im[6];  // NOLINT(runtime/arrays)
  int d_col_size[6];  // NOLINT(runtime/arrays)
  int d_col_iter[6];  // NOLINT(runtime/arrays)
  int d_col_start[6];  // NOLINT(runtime/arrays)
  int d_col_end[6];  // NOLINT(runtime/arrays)
  int d_ext_patch[6];  // NOLINT(runtime/arrays)
  int d_idx[6];  // NOLINT(runtime/arrays)

  for (int i = num_axes - 1; i >= 0; --i) {
    d_ext_patch[i] = (kernel_shape[i] - 1) * kstride[i] + 1;
    d_col_size[i] = (im_shape[i + 1] + 2 * pad[i] - d_ext_patch[i]) / stride[i]
        + 1;
  }

  CUDA_KERNEL_LOOP(index, n) {
    // Initialize channel_in, computed in the loop below, with intermediate
    // computations used to compute the spatial indices.
    int channel_im = index;
    // Calculate d_im (image dimensions).
    for (int i = num_axes - 1; i >= 0; --i) {
      d_im[i] = channel_im % im_shape[i + 1] + pad[i];
      channel_im /= im_shape[i + 1];
    }
    // Calculate col start/end indices.
    bool done = false;
    for (int i = 0; i < num_axes; ++i) {
      // Old:
      /*d_col_start[i] = d_col_iter[i] =
       (d_im[i] < kernel_shape[i]) ?
       0 : (d_im[i] - kernel_shape[i]) / stride[i] + 1;
       d_col_end[i] = min(d_im[i] / stride[i] + 1, col_shape[i + 1]);*/
      // New:
      d_col_start[i] =
          (d_im[i] < d_ext_patch[i]) ?
              d_im[i] % kstride[i] : (d_im[i] - d_ext_patch[i]) + 1;
      d_col_iter[i] = d_col_start[i];
      d_idx[i] = (d_im[i] - d_col_start[i]) / kstride[i];
      d_col_end[i] =
          (d_im[i] >= d_col_size[i]) ?
              (d_col_size[i] - 1)
                  - ((d_col_size[i] - 1) - d_col_start[i]) % kstride[i] :
              d_im[i];
      if (d_col_start[i] > d_col_end[i]) {
        // Skip computation if the dimension is 0 at any spatial axis --
        // final val will be 0.
        data_im[index] = 0;
        done = true;
        break;  // for (int i = 0; i < num_axes; ++i)
      }
    }
    if (done) {
      continue;  // CUDA_KERNEL_LOOP(index, n)
    }
    // Loop over the col to compute the output val.
    Dtype val = 0;
    bool incremented = true;
    do {
      // Compute the final offset.
      int final_offset = 0;
      int coeff_prod = 1;
      for (int i = num_axes - 1; i >= 0; --i) {
        final_offset += d_col_iter[i] * coeff_prod;
        coeff_prod *= d_col_size[i];
      }
      for (int i = num_axes - 1; i >= 0; --i) {
        final_offset += d_idx[i] * coeff_prod;
        coeff_prod *= kernel_shape[i];
      }
      final_offset += channel_im * coeff_prod;
      val += data_col[final_offset];
      incremented = false;
      for (int i = num_axes - 1; i >= 0; --i) {
        if (d_col_iter[i] > d_col_end[i] - kstride[i]) {
          d_col_iter[i] = d_col_start[i];
          d_idx[i] = (d_im[i] - d_col_start[i]) / kstride[i];
        } else {  // d_col_iter[i] <= d_max - kstride[1]
          d_col_iter[i] += kstride[i];
          --d_idx[i];
          incremented = true;
          break;  // for (int i = num_axes - 1; i >= 0; --i)
        }
      }  // for (int i = num_axes - 1; i >= 0; --i)
    } while (incremented);
    data_im[index] = val;
  }  // CUDA_KERNEL_LOOP(index, n)
}

template<typename Dtype>
void im2col_ndsk_gpu(const Dtype* data_im, const int num_spatial_axes,
                     const int num_kernels, const int* im_shape,
                     const int* col_shape, const int* kernel_shape,
                     const int* pad, const int* stride, const int* kstride,
                     Dtype* data_col) {
  im2col_ndsk_gpu_kernel<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
  CUDA_KERNEL(CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS) (
      num_kernels, num_spatial_axes, data_im, im_shape, col_shape,
      kernel_shape, pad, stride, kstride, data_col);
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void im2col_ndsk_gpu(const float* data_im, const int num_spatial_axes,
                              const int num_kernels, const int* im_shape,
                              const int* col_shape, const int* kernel_shape,
                              const int* pad, const int* stride,
                              const int* kstride, float* data_col);
template void im2col_ndsk_gpu(const double* data_im, const int num_spatial_axes,
                              const int num_kernels, const int* im_shape,
                              const int* col_shape, const int* kernel_shape,
                              const int* pad, const int* stride,
                              const int* kstride, double* data_col);

template<typename Dtype>
void col2im_ndsk_gpu(const Dtype* data_col, const int num_spatial_axes,
                     const int im_size, const int* im_shape,
                     const int* col_shape, const int* kernel_shape,
                     const int* pad, const int* stride, const int* kstride,
                     Dtype* data_im) {
  col2im_ndsk_gpu_kernel<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
  CUDA_KERNEL(CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS)(
      im_size, num_spatial_axes, data_col, im_shape, col_shape,
      kernel_shape, pad, stride, kstride, data_im);
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void col2im_ndsk_gpu(const float* data_col, const int num_axes,
                              const int im_size, const int* im_shape,
                              const int* col_shape, const int* kernel_shape,
                              const int* pad, const int* stride,
                              const int* kstride, float* data_im);
template void col2im_ndsk_gpu(const double* data_col, const int num_axes,
                              const int im_size, const int* im_shape,
                              const int* col_shape, const int* kernel_shape,
                              const int* pad, const int* stride,
                              const int* kstride, double* data_im);

template<typename Dtype>
__global__ void im2col_nd_gpu_kernel(const int n, const int num_axes,
                                     const Dtype* data_im, const int* im_shape,
                                     const int* col_shape,
                                     const int* kernel_shape, const int* pad,
                                     const int* stride, Dtype* data_col) {
  int d_temp[6];  // NOLINT(runtime/arrays)
  int d_iter[6];  // NOLINT(runtime/arrays)
  int i;
  CUDA_KERNEL_LOOP(index, n) {
    // Initialize channel_in, computed in the loop below, with intermediate
    // computations used to compute the spatial indices.
    int channel_in = index;
    int channel_out = 1;
    for (i = num_axes - 1; i >= 0; --i) {
      d_temp[i] = channel_in % col_shape[i + 1];
      channel_in /= col_shape[i + 1];
      channel_out *= kernel_shape[i];
    }
    channel_out *= channel_in;
    int data_col_inc = 1;
    for (i = 0; i < num_axes; ++i) {
      channel_out *= col_shape[i + 1];
      channel_out += d_temp[i];
      d_temp[i] = d_temp[i] * stride[i] - pad[i];
      channel_in *= im_shape[i + 1];
      channel_in += d_temp[i];
      data_col_inc *= col_shape[i + 1];
      d_iter[i] = 0;
    }
    Dtype* data_col_ptr = data_col + channel_out;
    const Dtype* data_im_ptr = data_im + channel_in;
    bool incremented;
    do {
      bool in_range = true;
      for (i = 0; i < num_axes; ++i) {
        const int d_iter_im = d_iter[i] + d_temp[i];
        in_range &= d_iter_im >= 0 && d_iter_im < im_shape[i + 1];
        if (!in_range) {
          break;
        }
      }
      if (in_range) {
        int data_im_offset = d_iter[0];
        for (i = 1; i < num_axes; ++i) {
          data_im_offset *= im_shape[i + 1];
          data_im_offset += d_iter[i];
        }
        *data_col_ptr = data_im_ptr[data_im_offset];
      } else {
        *data_col_ptr = 0;
      }
      data_col_ptr += data_col_inc;
      incremented = false;
      for (i = num_axes - 1; i >= 0; --i) {
        const int d_max = kernel_shape[i];
        if (d_iter[i] == d_max - 1) {
          d_iter[i] = 0;
        } else {  // d_iter[i] < d_max - 1
          ++d_iter[i];
          incremented = true;
          break;
        }
      }  // for (int i = num_axes - 1; i >= 0; --i)
    } while (incremented);  // do
  }  // CUDA_KERNEL_LOOP(index, n)
}

template<typename Dtype>
void im2col_nd_gpu(const Dtype* data_im, const int num_spatial_axes,
                   const int num_kernels, const int* im_shape,
                   const int* col_shape, const int* kernel_shape,
                   const int* pad, const int* stride, Dtype* data_col) {
  im2col_nd_gpu_kernel<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
  CUDA_KERNEL(CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS)(
      num_kernels, num_spatial_axes, data_im, im_shape, col_shape,
      kernel_shape, pad, stride, data_col);
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void im2col_nd_gpu<float>(const float* data_im,
                                   const int num_spatial_axes,
                                   const int col_size, const int* im_shape,
                                   const int* col_shape,
                                   const int* kernel_shape, const int* pad,
                                   const int* stride, float* data_col);
template void im2col_nd_gpu<double>(const double* data_im,
                                    const int num_spatial_axes,
                                    const int col_size, const int* im_shape,
                                    const int* col_shape,
                                    const int* kernel_shape, const int* pad,
                                    const int* stride, double* data_col);

template<typename Dtype>
__global__ void col2im_nd_gpu_kernel(const int n, const int num_axes,
                                     const Dtype* data_col, const int* im_shape,
                                     const int* col_shape,
                                     const int* kernel_shape, const int* pad,
                                     const int* stride, Dtype* data_im) {
  int d_im[6];  // NOLINT(runtime/arrays)
  int d_col_iter[6];  // NOLINT(runtime/arrays)
  int d_col_start[6];  // NOLINT(runtime/arrays)
  int d_col_end[6];  // NOLINT(runtime/arrays)
  CUDA_KERNEL_LOOP(index, n) {
    // Initialize channel_in, computed in the loop below, with intermediate
    // computations used to compute the spatial indices.
    int channel_im = index;
    // Calculate d_im (image dimensions).
    for (int i = num_axes - 1; i >= 0; --i) {
      d_im[i] = channel_im % im_shape[i + 1] + pad[i];
      channel_im /= im_shape[i + 1];
    }
    // Calculate col start/end indices.
    bool done = false;
    for (int i = 0; i < num_axes; ++i) {
      d_col_start[i] = d_col_iter[i] =
          (d_im[i] < kernel_shape[i]) ?
              0 : (d_im[i] - kernel_shape[i]) / stride[i] + 1;
      d_col_end[i] = min(d_im[i] / stride[i] + 1, col_shape[i + 1]);
      if (d_col_start[i] >= d_col_end[i]) {
        // Skip computation if the dimension is 0 at any spatial axis --
        // final val will be 0.
        data_im[index] = 0;
        done = true;
        break;  // for (int i = 0; i < num_axes; ++i)
      }
    }
    if (done) {
      continue;  // CUDA_KERNEL_LOOP(index, n)
    }
    // Loop over the col to compute the output val.
    Dtype val = 0;
    bool incremented = true;
    do {
      // Compute the final offset.
      int final_offset = 0;
      int kernel_shape_prod = 1;
      for (int i = num_axes - 1; i >= 0; --i) {
        final_offset += (d_im[i] - d_col_iter[i] * stride[i])
            * kernel_shape_prod;
        kernel_shape_prod *= kernel_shape[i];
      }
      final_offset += kernel_shape_prod * channel_im;
      for (int i = 0; i < num_axes; ++i) {
        final_offset *= col_shape[i + 1];
        final_offset += d_col_iter[i];
      }
      val += data_col[final_offset];
      incremented = false;
      for (int i = num_axes - 1; i >= 0; --i) {
        const int d_max = d_col_end[i];
        if (d_col_iter[i] == d_max - 1) {
          d_col_iter[i] = d_col_start[i];
        } else {  // d_col_iter[i] < d_max - 1
          ++d_col_iter[i];
          incremented = true;
          break;  // for (int i = num_axes - 1; i >= 0; --i)
        }
      }  // for (int i = num_axes - 1; i >= 0; --i)
    } while (incremented);
    data_im[index] = val;
  }  // CUDA_KERNEL_LOOP(index, n)
}

template<typename Dtype>
void col2im_nd_gpu(const Dtype* data_col, const int num_spatial_axes,
                   const int im_size, const int* im_shape, const int* col_shape,
                   const int* kernel_shape, const int* pad, const int* stride,
                   Dtype* data_im) {
  col2im_nd_gpu_kernel<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
  CUDA_KERNEL(CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS)(
      im_size, num_spatial_axes, data_col, im_shape, col_shape,
      kernel_shape, pad, stride, data_im);
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void col2im_nd_gpu<float>(const float* data_col,
                                   const int num_spatial_axes,
                                   const int im_size, const int* im_shape,
                                   const int* col_shape,
                                   const int* kernel_shape, const int* pad,
                                   const int* stride, float* data_im);
template void col2im_nd_gpu<double>(const double* data_col,
                                    const int num_spatial_axes,
                                    const int im_size, const int* im_shape,
                                    const int* col_shape,
                                    const int* kernel_shape, const int* pad,
                                    const int* stride, double* data_im);

#endif  // USE_CUDA
}  // namespace caffe
